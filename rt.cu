#include "hip/hip_runtime.h"
#include "consts.hpp"
#include "linalg.hpp"
#include "rt.hpp"

mat4 *g_proj = NULL;
uint32_t *g_buf = NULL;
float3 *g_light = NULL;

__device__ ray camera_ray(int x, int y, mat4 *proj) {
	float3 dir = {0, 0, 1};
	dir.x = (x / (WIDTH / 2.0) - 1) / (*proj)[0][0];
	dir.y = (y / (HEIGHT / 2.0) - 1) / (*proj)[1][1];

	ray r = {{0, 0, 0}, {dir.x, dir.y, dir.z}};
	return r;
}

__device__ float3 raycast(ray r, int numTriangles, float3 *verticies, uint3 *triangles, float3 *light, int depth) {
	float3 res = {0, 0, 0};
	// Search for the closest
	int closest = -1;
	float closestDist = INFINITY;
	float3 closestPoint;
	float3 normal;
	for (int i = 0; i < numTriangles; i++) {
		float3 *a = verticies + triangles[i].x;
		float3 *b = verticies + triangles[i].y;
		float3 *c = verticies + triangles[i].z;

		plane p;
		float3 ab, ac;
		vec3_sub(&ab, b, a);
		vec3_sub(&ac, c, a);
		vec3_cross((float3 *)&p, &ab, &ac);
		vec3_normalize((float3 *)&p, (float3 *)&p);
		p.w = -vec3_dot((float3 *)&p, a);

		float3 point;
		if (intersect_plane_line(&point, &p, &r)) {
			if (point.z > FAR || point.z < NEAR)
				continue;
			if (point_in_triangle(&point, a, b, c)) {
				float dist = sqrtf(point.x * point.x + point.y * point.y + point.z * point.z);
				if (dist < closestDist) {
					closest = i;
					closestDist = dist;
					closestPoint = point;
					normal = *(float3 *)&p;
				}
			}
		}
	}
	if (depth == 0) {
		if (closest != -1) {
			float3 lightDir;
			vec3_sub(&lightDir, light, &closestPoint);
			vec3_normalize(&lightDir, &lightDir);
			float lightDot = fabsf(vec3_dot(&lightDir, &normal));
			if (lightDot < 0)
				lightDot = 0;
			res.x = lightDot;
			res.y = lightDot;
			res.z = lightDot;
		} else {
			res.x = 0;
			res.y = 0;
			res.z = 0;
		}
	} else {
		// Send out rays in many directions
	}
	return res;
}

__global__ void raytrace(int numTriangles, uint32_t *buf, float3 *verticies, uint3 *triangles, mat4 *proj, float3 *light) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	ray r = camera_ray(x, HEIGHT - 1 - y, proj);

	float3 res = raycast(r, numTriangles, verticies, triangles, light, 0);
	uchar3 color = {(uint8_t)(res.x * 255), (uint8_t)(res.y * 255), (uint8_t)(res.z * 255)};

	buf[y * WIDTH + x] = (color.x << 16) | (color.y << 8) | color.z;

	return;
}

void raytrace_render(uint32_t *out, float3 *verts, uint3 *tris, int numTris, mat4 arg_proj, float3 *light) {
	if (g_buf == NULL) {
		hipMalloc(&g_buf, sizeof(uint32_t) * WIDTH * HEIGHT);
		hipMalloc(&g_proj, sizeof(mat4));
		hipMemcpy(*g_proj, arg_proj, sizeof(mat4), hipMemcpyHostToDevice);
		hipMalloc(&g_light, sizeof(float3));
	}

	float3 *verticies;
	uint3 *triangles;

	hipMalloc(&verticies, sizeof(float3) * numTris * 3);
	hipMalloc(&triangles, sizeof(uint3) * numTris);
	hipMemcpy(verticies, verts, sizeof(float3) * numTris * 3, hipMemcpyHostToDevice);
	hipMemcpy(triangles, tris, sizeof(uint3) * numTris, hipMemcpyHostToDevice);
	hipMemcpy(g_light, light, sizeof(float3), hipMemcpyHostToDevice);

	dim3 block(8, 8);
	dim3 grid(WIDTH / block.x, HEIGHT / block.y);

	raytrace<<<grid, block>>>(numTris, g_buf, verticies, triangles, g_proj, g_light);

	hipError_t err = hipDeviceSynchronize();
	if (err != hipSuccess) {
		fprintf(stderr, "Failed to synchronize: %s\n", hipGetErrorString(err));
		exit(1);
	}
	hipMemcpy(out, g_buf, sizeof(uint32_t) * WIDTH * HEIGHT, hipMemcpyDeviceToHost);

	hipFree(verticies);
	hipFree(triangles);
}
