#include "hip/hip_runtime.h"
#include "consts.hpp"
#include "linalg.hpp"
#include "rt.hpp"

mat4 *g_proj = NULL;

uint32_t *g_buf = NULL;
float3 *g_verticies = NULL;
uint3 *g_triangles = NULL;

__device__ ray camera_ray(int x, int y, mat4 *proj) {
	float3 dir = {0, 0, 1};
	dir.x = (x / (WIDTH / 2) - 1) / (*proj)[0][0];
	dir.y = (y / (HEIGHT / 2) - 1) / (*proj)[1][1];
	vec3_normalize(&dir, &dir);

	ray r = {{0, 0, 0}, {dir.x, dir.y, dir.z}};
	return r;
}

__device__ float3 raycast(ray r, int depth) {
	/// TODO: Implement
	float3 res = {0, 0, 0};
	if (depth == 0) {
		// Go straight to the light
	} else {
		// Send out rays in many directions
	}
	/// TODO: Remove dummy code when done implementing
	res = {0.5, 0.5, 0.5};
	return res;
}

__global__ void raytrace(int numTriangles, uint32_t *buf, float3 *verticies, uint3 *triangles, mat4 *proj) {
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	ray r = camera_ray(x, y, proj);

	float3 res = raycast(r, 0);
	uchar3 color = {(uint8_t)(res.x * 255), (uint8_t)(res.y * 255), (uint8_t)(res.z * 255)};

	buf[y * WIDTH + x] = (color.x << 16) | (color.y << 8) | color.z;

	return;
}

void raytrace_render(uint32_t *out, float3 *verts, uint3 *tris, mat4 arg_proj, int numTris) {
	if (g_buf == NULL) {
		hipMalloc(&g_buf, sizeof(uint32_t) * WIDTH * HEIGHT);
		hipMalloc(&g_proj, sizeof(mat4));
		hipMemcpy(g_proj, &arg_proj, sizeof(mat4), hipMemcpyHostToDevice);
	}

	hipMalloc(&g_verticies, sizeof(float3) * numTris * 3);
	hipMalloc(&g_triangles, sizeof(uint3) * numTris);
	hipMemcpy(g_verticies, verts, sizeof(float3) * numTris * 3, hipMemcpyHostToDevice);
	hipMemcpy(g_triangles, tris, sizeof(uint3) * numTris, hipMemcpyHostToDevice);

	dim3 block(16, 16);
	dim3 grid(WIDTH / block.x, HEIGHT / block.y);

	raytrace<<<grid, block>>>(numTris, g_buf, g_verticies, g_triangles, g_proj);

	hipError_t err = hipDeviceSynchronize();
	if (err != hipSuccess) {
		fprintf(stderr, "Failed to synchronize: %s\n", hipGetErrorString(err));
		exit(1);
	}
	hipMemcpy(out, g_buf, sizeof(uint32_t) * WIDTH * HEIGHT, hipMemcpyDeviceToHost);

	hipFree(g_verticies);
	hipFree(g_triangles);
}
