#include "hip/hip_runtime.h"
#include "vec.hpp"

special float vec3_dot(const float3 *a, const float3 *b) {
	return a->x * b->x + a->y * b->y + a->z * b->z;
}

special void vec3_cross(float3 *out, const float3 *a, const float3 *b) {
	out->x = a->y * b->z - a->z * b->y;
	out->y = a->z * b->x - a->x * b->z;
	out->z = a->x * b->y - a->y * b->x;
}

special float vec3_length(const float3 *a) {
	return sqrtf(vec3_dot(a, a));
}

special void vec3_normalize(float3 *out, const float3 *a) {
	float len = vec3_length(a);
	if (__builtin_expect(len == 0, false)) {
		out->x = 0;
		out->y = 0;
		out->z = 0;
		return;
	}
	out->x = a->x / len;
	out->y = a->y / len;
	out->z = a->z / len;
}

special void vec3_add(float3 *out, const float3 *a, const float3 *b) {
	out->x = a->x + b->x;
	out->y = a->y + b->y;
	out->z = a->z + b->z;
}

special void vec3_sub(float3 *out, const float3 *a, const float3 *b) {
	out->x = a->x - b->x;
	out->y = a->y - b->y;
	out->z = a->z - b->z;
}

special void vec3_scale(float3 *out, const float3 *a, const float s) {
	out->x = a->x * s;
	out->y = a->y * s;
	out->z = a->z * s;
}

special void vec3_negate(float3 *out, const float3 *a) {
	out->x = -a->x;
	out->y = -a->y;
	out->z = -a->z;
}

special void vec3_tovec4(float4 *out, const float3 *a) {
	out->x = a->x;
	out->y = a->y;
	out->z = a->z;
	out->w = 1;
}

special void vec3_lerp_x(float3 *out, const float3 *a, const float3 *b, const float x) {
	out->x = x;
	out->y = a->y + (b->y - a->y) * (x - a->x) / (b->x - a->x);
	out->z = a->z + (b->z - a->z) * (x - a->x) / (b->x - a->x);
}

special void vec3_lerp_y(float3 *out, const float3 *a, const float3 *b, const float y) {
	out->x = a->x + (b->x - a->x) * (y - a->y) / (b->y - a->y);
	out->y = y;
	out->z = a->z + (b->z - a->z) * (y - a->y) / (b->y - a->y);
}

special void vec3_lerp_z(float3 *out, const float3 *a, const float3 *b, const float z) {
	out->x = a->x + (b->x - a->x) * (z - a->z) / (b->z - a->z);
	out->y = a->y + (b->y - a->y) * (z - a->z) / (b->z - a->z);
	out->z = z;
}
