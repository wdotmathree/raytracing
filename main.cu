#include "hip/hip_runtime.h"
#include "consts.hpp"
#include "linalg.hpp"
#include "mat.hpp"
#include "rt.hpp"
#include "transform.hpp"
#include "vec.hpp"

#include <stdio.h>
#include <sys/mman.h>

#include <vector>

const float3 g_light = {0, 0, -10};
float3 camera = {2, 1, -5};
float2 look = {0, 0};

int numVerticies;
int numTriangles;
float3 *verticies;
float3 *transVerts;
uint3 *triangles;

uint32_t *frame_buffer;

void sort(uint3 *arr, int n) {
	if (n < 2)
		return;
	float pivot = verticies[arr[n / 2].x].z + verticies[arr[n / 2].y].z + verticies[arr[n / 2].z].z;
	int l = 0;
	int r = n - 1;
	while (l <= r) {
		if (verticies[arr[l].x].z + verticies[arr[l].y].z + verticies[arr[l].z].z < pivot) {
			l++;
			continue;
		}
		if (verticies[arr[r].x].z + verticies[arr[r].y].z + verticies[arr[r].z].z > pivot) {
			r--;
			continue;
		}
		uint3 tmp = arr[l];
		arr[l] = arr[r];
		arr[r] = tmp;
		l++;
		r--;
	}
	sort(arr, r + 1);
	sort(arr + l, n - l);
}

int parseFile() {
	// Read in the data
	FILE *fp = fopen("thing.obj", "r");
	if (!fp) {
		fprintf(stderr, "Failed to open file\n");
		return 1;
	}
	fseek(fp, 0, SEEK_END);
	char *map = (char *)mmap(NULL, ftell(fp) + 1, PROT_READ, MAP_PRIVATE, fp->_fileno, 0);
	char *line = map;
	while (line) {
		if (line[0] == 'v' && line[1] == ' ') {
			numVerticies++;
		} else if (line[0] == 'f' && line[1] == ' ') {
			numTriangles++;
		}
		line = strchr(line, '\n');
		if (line)
			line++;
	}
	verticies = (float3 *)malloc(numVerticies * sizeof(float3));
	triangles = (uint3 *)malloc(numTriangles * sizeof(uint3));
	transVerts = (float3 *)malloc(numVerticies * sizeof(float3));
	line = map;
	int v = 0;
	int t = 0;
	// float minx = 0;
	// float miny = 0;
	// float minz = 0;
	// float maxx = 0;
	// float maxy = 0;
	// float maxz = 0;
	while (line) {
		if (line[0] == 'v' && line[1] == ' ') {
			float3 *vert = verticies + v;
			sscanf(line, "v %f %f %f", &vert->x, &vert->y, &vert->z);
			// minx = fminf(minx, vert->x);
			// miny = fminf(miny, vert->y);
			// minz = fminf(minz, vert->z);
			// maxx = fmaxf(maxx, vert->x);
			// maxy = fmaxf(maxy, vert->y);
			// maxz = fmaxf(maxz, vert->z);
			v++;
		} else if (line[0] == 'f' && line[1] == ' ') {
			uint3 *tri = triangles + t;
			sscanf(line, "f %d %d %d", &tri->x, &tri->y, &tri->z);
			tri->x--;
			tri->y--;
			tri->z--;
			t++;
		}
		line = strchr(line, '\n');
		if (line)
			line++;
	}
	// float scale = fmaxf(fmaxf(maxx - minx, maxy - miny), maxz - minz) / 2;
	// for (int i = 0; i < numVerticies; i++) {
	// 	float3 *vert = verticies + i;
	// 	vert->x -= (maxx + minx) / 2;
	// 	vert->y -= (maxy + miny) / 2;
	// 	vert->z -= (maxz + minz) / 2;
	// 	vert->x /= scale;
	// 	vert->y /= scale;
	// 	vert->z /= scale;
	// }
	munmap(map, ftell(fp) + 1);
	fclose(fp);

	// Sort the triangles
	sort(triangles, numTriangles);

	return 0;
}

mat4 proj;

static plane top, bottom, left, right, near, far;
static plane *planes[] = {&top, &bottom, &left, &right, &near, &far};

// top, bottom, left, right, near, far
bool is_in(float3 *v, int selector) {
	plane *p = planes[selector];
	switch (selector) {
	case 0:
	case 1:
		return v->y * p->y > (NEAR - v->z) * p->z;
	case 2:
	case 3:
		return v->x * p->x > (NEAR - v->z) * p->z;
	case 4:
		return v->z > -p->w;
	case 5:
		return v->z < -p->w;
	}
	exit(1);
}

void render() {
	// Transform the verticies
	mat4 t1, t2, rot;
	rotateY(t1, -look.x);
	rotateX(t2, look.y);
	mat4_mul(rot, t1, t2);
	translate(t1, -camera.x, -camera.y, -camera.z);
	mat4_mul(t2, rot, t1);
	for (int i = 0; i < numVerticies; i++) {
		float4 v;
		vec3_tovec4(&v, &verticies[i]);
		mat4_mul_vec(&v, t2, &v);
		memcpy(&transVerts[i], &v, sizeof(float3));
	}
	// Transform the light source
	float4 tmp_light;
	vec3_tovec4(&tmp_light, &g_light);
	mat4_mul_vec(&tmp_light, t2, &tmp_light);
	float3 light;
	light.x = tmp_light.x;
	light.y = tmp_light.y;
	light.z = tmp_light.z;
	raytrace_render(frame_buffer, transVerts, triangles, numTriangles, proj, (float3 *)&light);
}

int main() {
	top = (plane){0, -tanf(FOV / 2 * M_PI / 180), FAR / (FAR - NEAR), -(FAR * NEAR) / (FAR - NEAR)};
	bottom = (plane){0, tanf(FOV / 2 * M_PI / 180), FAR / (FAR - NEAR), -(FAR * NEAR) / (FAR - NEAR)};
	left = (plane){ASPECT * tanf(FOV / 2 * M_PI / 180), 0, FAR / (FAR - NEAR), -(FAR * NEAR) / (FAR - NEAR)};
	right = (plane){-ASPECT * tanf(FOV / 2 * M_PI / 180), 0, FAR / (FAR - NEAR), -(FAR * NEAR) / (FAR - NEAR)};
	near = (plane){0, 0, 1, -NEAR};
	far = (plane){0, 0, 1, -FAR};
	project(proj, FOV, ASPECT, NEAR, FAR);
	if (parseFile())
		return 1;
	frame_buffer = (uint32_t *)calloc(WIDTH * HEIGHT, sizeof(uint32_t));
	render();
	return 0;
}
