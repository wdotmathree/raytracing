#include "hip/hip_runtime.h"
#include "consts.hpp"
#include "linalg.hpp"
#include "mat.hpp"
#include "rt.hpp"
#include "transform.hpp"
#include "vec.hpp"

#include <SDL.h>
#include <pthread.h>
#include <stdio.h>
#include <sys/mman.h>
#include <time.h>

#include <vector>

const int FPS = 15;
const float3 light = {0, 0, -10};
float3 camera = {2, 1, -5};
float2 look = {0, 0};

bool raytrace = false;

int numVerticies;
int numTriangles;
float3 *verticies;
float3 *transVerts;
uint3 *triangles;

uint32_t *frame_buffer;
uint32_t *middle_buffer;
float *z_buffer;

void fillTriangle(int lum, float3 *a, float3 *b, float3 *c) {
	float3 left, mid, right;
	if (a->x < b->x) {
		if (a->x < c->x) {
			left = *a;
			if (b->x < c->x) {
				mid = *b;
				right = *c;
			} else {
				mid = *c;
				right = *b;
			}
		} else {
			left = *c;
			mid = *a;
			right = *b;
		}
	} else {
		if (b->x < c->x) {
			left = *b;
			if (a->x < c->x) {
				mid = *a;
				right = *c;
			} else {
				mid = *c;
				right = *a;
			}
		} else {
			left = *c;
			mid = *b;
			right = *a;
		}
	}
	float m1 = (left.y - right.y) / (left.x - right.x);
	float m2 = (left.y - mid.y) / (left.x - mid.x);
	float m3 = (mid.y - right.y) / (mid.x - right.x);
	float mz1 = (left.z - right.z) / (left.x - right.x);
	float mz2 = (left.z - mid.z) / (left.x - mid.x);
	float mz3 = (mid.z - right.z) / (mid.x - right.x);
	float b1 = left.y - m1 * left.x;
	float b2 = left.y - m2 * left.x;
	float b3 = mid.y - m3 * mid.x;
	float bz1 = left.z - mz1 * left.x;
	float bz2 = left.z - mz2 * left.x;
	float bz3 = mid.z - mz3 * mid.x;
	for (int x = ceilf(left.x); x <= floorf(mid.x); x++) {
		float y1 = m1 * x + b1;
		float y2 = m2 * x + b2;
		if (y1 > y2) {
			float t = y1;
			y1 = y2;
			y2 = t;
		}
		float z1 = mz1 * x + bz1;
		float z2 = mz2 * x + bz2;
		// if (z1 < z_buffer[(int)ceilf(y1) * WIDTH + x]) {
		// 	z_buffer[(int)ceilf(y1) * WIDTH + x] = z1;
		// 	frame_buffer[(int)ceilf(y1) * WIDTH + x] = 0;
		// }
		// if (z2 < z_buffer[(int)floorf(y2) * WIDTH + x]) {
		// 	z_buffer[(int)floorf(y2) * WIDTH + x] = z2;
		// 	frame_buffer[(int)floorf(y2) * WIDTH + x] = 0;
		// }
		for (int y = ceilf(y1); y <= floorf(y2); y++) {
			float z = z1 + (z2 - z1) * (y - y1) / (y2 - y1);
			if (z < z_buffer[y * WIDTH + x]) {
				z_buffer[y * WIDTH + x] = z;
				frame_buffer[y * WIDTH + x] = (lum << 16) | (lum << 8) | lum;
			}
		}
	}
	for (int x = ceilf(mid.x); x <= floorf(right.x); x++) {
		float y1 = m1 * x + b1;
		float y2 = m3 * x + b3;
		if (y1 > y2) {
			float t = y1;
			y1 = y2;
			y2 = t;
		}
		float z1 = mz1 * x + bz1;
		float z2 = mz3 * x + bz3;
		// if (z1 < z_buffer[(int)ceilf(y1) * WIDTH + x]) {
		// 	z_buffer[(int)ceilf(y1) * WIDTH + x] = z1;
		// 	frame_buffer[(int)ceilf(y1) * WIDTH + x] = 0;
		// }
		// if (z2 < z_buffer[(int)floorf(y2) * WIDTH + x]) {
		// 	z_buffer[(int)floorf(y2) * WIDTH + x] = z2;
		// 	frame_buffer[(int)floorf(y2) * WIDTH + x] = 0;
		// }
		for (int y = ceilf(y1); y <= floorf(y2); y++) {
			float z = z1 + (z2 - z1) * (y - y1) / (y2 - y1);
			if (z < z_buffer[y * WIDTH + x]) {
				z_buffer[y * WIDTH + x] = z;
				frame_buffer[y * WIDTH + x] = (lum << 16) | (lum << 8) | lum;
			}
		}
	}
}

void centroid(float3 *centroid, float3 *a, float3 *b, float3 *c) {
	centroid->x = (a->x + b->x + c->x) / 3;
	centroid->y = (a->y + b->y + c->y) / 3;
	centroid->z = (a->z + b->z + c->z) / 3;
}

void sort(uint3 *arr, int n) {
	if (n < 2)
		return;
	float pivot = verticies[arr[n / 2].x].z + verticies[arr[n / 2].y].z + verticies[arr[n / 2].z].z;
	int l = 0;
	int r = n - 1;
	while (l <= r) {
		if (verticies[arr[l].x].z + verticies[arr[l].y].z + verticies[arr[l].z].z < pivot) {
			l++;
			continue;
		}
		if (verticies[arr[r].x].z + verticies[arr[r].y].z + verticies[arr[r].z].z > pivot) {
			r--;
			continue;
		}
		uint3 tmp = arr[l];
		arr[l] = arr[r];
		arr[r] = tmp;
		l++;
		r--;
	}
	sort(arr, r + 1);
	sort(arr + l, n - l);
}

int parseFile() {
	// Read in the data
	FILE *fp = fopen("thing.obj", "r");
	if (!fp) {
		fprintf(stderr, "Failed to open file\n");
		return 1;
	}
	fseek(fp, 0, SEEK_END);
	char *map = (char *)mmap(NULL, ftell(fp) + 1, PROT_READ, MAP_PRIVATE, fp->_fileno, 0);
	char *line = map;
	while (line) {
		if (line[0] == 'v' && line[1] == ' ') {
			numVerticies++;
		} else if (line[0] == 'f' && line[1] == ' ') {
			numTriangles++;
		}
		line = strchr(line, '\n');
		if (line)
			line++;
	}
	verticies = (float3 *)malloc(numVerticies * sizeof(float3));
	triangles = (uint3 *)malloc(numTriangles * sizeof(uint3));
	transVerts = (float3 *)malloc(numVerticies * sizeof(float3));
	line = map;
	int v = 0;
	int t = 0;
	// float minx = 0;
	// float miny = 0;
	// float minz = 0;
	// float maxx = 0;
	// float maxy = 0;
	// float maxz = 0;
	while (line) {
		if (line[0] == 'v' && line[1] == ' ') {
			float3 *vert = verticies + v;
			sscanf(line, "v %f %f %f", &vert->x, &vert->y, &vert->z);
			// minx = fminf(minx, vert->x);
			// miny = fminf(miny, vert->y);
			// minz = fminf(minz, vert->z);
			// maxx = fmaxf(maxx, vert->x);
			// maxy = fmaxf(maxy, vert->y);
			// maxz = fmaxf(maxz, vert->z);
			v++;
		} else if (line[0] == 'f' && line[1] == ' ') {
			uint3 *tri = triangles + t;
			sscanf(line, "f %d %d %d", &tri->x, &tri->y, &tri->z);
			tri->x--;
			tri->y--;
			tri->z--;
			t++;
		}
		line = strchr(line, '\n');
		if (line)
			line++;
	}
	// float scale = fmaxf(fmaxf(maxx - minx, maxy - miny), maxz - minz) / 2;
	// for (int i = 0; i < numVerticies; i++) {
	// 	float3 *vert = verticies + i;
	// 	vert->x -= (maxx + minx) / 2;
	// 	vert->y -= (maxy + miny) / 2;
	// 	vert->z -= (maxz + minz) / 2;
	// 	vert->x /= scale;
	// 	vert->y /= scale;
	// 	vert->z /= scale;
	// }
	munmap(map, ftell(fp) + 1);
	fclose(fp);

	// Sort the triangles
	sort(triangles, numTriangles);

	return 0;
}

int frame = 0;
long long prevtime = 0;
SDL_Window *w;
SDL_Surface *s;

mat4 proj;
pthread_t t;

// Sync paint with render
pthread_mutex_t paint_mutex = PTHREAD_MUTEX_INITIALIZER;

static plane top, bottom, left, right, near, far;
static plane *planes[] = {&top, &bottom, &left, &right, &near, &far};

// top, bottom, left, right, near, far
bool is_in(float3 *v, int selector) {
	plane *p = planes[selector];
	switch (selector) {
	case 0:
	case 1:
		return v->y * p->y > (NEAR - v->z) * p->z;
	case 2:
	case 3:
		return v->x * p->x > (NEAR - v->z) * p->z;
	case 4:
		return v->z > -p->w;
	case 5:
		return v->z < -p->w;
	}
	exit(1);
}

std::vector<float3> *clip(float3 **verts, int numVerts) {
	// Prepare arrays
	std::vector<float3> *clipped = new std::vector<float3>();
	std::vector<float3> *tmp = new std::vector<float3>();
	std::vector<float3> inside = std::vector<float3>();
	clipped->reserve(numVerts * 2);
	tmp->reserve(numVerts * 2);
	for (int i = 0; i < numVerts; i++)
		tmp->push_back(*verts[i]);
	// Clip against each plane
	for (int i = 0; i < 6; i++) {
		clipped->clear();
		for (int j = 0; j < tmp->size(); j += 3) {
			inside.clear();
			float3 *a = &(*tmp)[j + 0];
			float3 *b = &(*tmp)[j + 1];
			float3 *c = &(*tmp)[j + 2];
			// Check if the triangle is inside the plane
			if (is_in(a, i) && is_in(b, i) && is_in(c, i)) {
				clipped->push_back(*a);
				clipped->push_back(*b);
				clipped->push_back(*c);
				continue;
			}
			// Check if the triangle is outside the plane
			if (!is_in(a, i) && !is_in(b, i) && !is_in(c, i))
				continue;
			// Check if the triangle intersects the plane
			float3 out1, out2;
			if (intersect_plane_triangle(&out1, &out2, planes[i], a, b, c) < 2) {
				clipped->push_back(*a);
				clipped->push_back(*b);
				clipped->push_back(*c);
				continue;
			}
			/// TODO: Fix this part
			bool used = false; // If we have used out1 yet
			int outside = 0; // How many verticies are outside the visible area
			// Inside are the points A,B,C (in order) and their replacements (out1, out2) if they
			// are outside
			if (is_in(a, i)) {
				inside.push_back(*a);
			} else {
				// Use out1 as a replacement for A
				inside.push_back(out1);
				used = true;
				outside++;
			}
			if (is_in(b, i)) {
				inside.push_back(*b);
			} else {
				// Use out2 if we have already used out1, otherwise use out1
				inside.push_back(used ? out2 : out1);
				used = true;
				outside++;
			}
			if (is_in(c, i)) {
				inside.push_back(*c);
			} else {
				inside.push_back(used ? out2 : out1);
				used = true;
				outside++;
			}
			if (outside == 1) {
				// Cutting a corner off, break into 2 triangles
				clipped->push_back(inside[0]);
				clipped->push_back(inside[1]);
				clipped->push_back(inside[2]);
				clipped->push_back(inside[2]);
				clipped->push_back(out2);
				clipped->push_back(inside[0]);
			} else {
				// Ctting a side off, still only 1 triangle
				clipped->push_back(inside[0]);
				clipped->push_back(inside[1]);
				clipped->push_back(inside[2]);
			}
		}
		// Swap the buffers
		std::swap(clipped, tmp);
	}
	delete tmp;
	return clipped;
}

void *renderLoop(void *args) {
	(void)args; // unused
	while (true) {
		// Wait for paint to finish
		pthread_mutex_lock(&paint_mutex);
		// Transform the verticies
		mat4 t1, t2, rot;
		rotateY(t1, -look.x);
		rotateX(t2, look.y);
		mat4_mul(rot, t1, t2);
		translate(t1, -camera.x, -camera.y, -camera.z);
		mat4_mul(t2, rot, t1);
		for (int i = 0; i < numVerticies; i++) {
			float4 v;
			vec3_tovec4(&v, &verticies[i]);
			mat4_mul_vec(&v, t2, &v);
			memcpy(&transVerts[i], &v, sizeof(float3));
		}
		if (raytrace) {
			raytrace_render(middle_buffer, transVerts, triangles, proj, numTriangles);
		} else {
			// Collect the verticies into triangles
			float3 **triVerticies = (float3 **)malloc(numTriangles * 3 * sizeof(float3 *));
			for (int i = 0; i < numTriangles; i++) {
				triVerticies[i * 3 + 0] = transVerts + triangles[i].x;
				triVerticies[i * 3 + 1] = transVerts + triangles[i].y;
				triVerticies[i * 3 + 2] = transVerts + triangles[i].z;
			}
			// Clip the triangles
			std::vector<float3> *clipped = clip(triVerticies, numTriangles * 3);
			// Project the verticies
			float3 *screenVerts = (float3 *)malloc(clipped->size() * sizeof(float3));
			for (int i = 0; i < clipped->size(); i++) {
				float4 a, b;
				vec3_tovec4(&a, &(*clipped)[i]);
				mat4_mul_vec(&b, proj, &a);
				// Make it to screen space
				to_screen(&screenVerts[i], (float3 *)&b);
			}
			// Draw the triangles
			memset(z_buffer, 0x7f, WIDTH * HEIGHT * sizeof(int));
			memset(frame_buffer, 0, WIDTH * HEIGHT * sizeof(uint32_t));
			for (int i = 0; i < clipped->size(); i += 3) {
				float3 *a = &(*clipped)[i + 0];
				float3 *b = &(*clipped)[i + 1];
				float3 *c = &(*clipped)[i + 2];
				// get the luminance
				float3 cent, norm, ab, ac, normcent;
				centroid(&cent, a, b, c);
				vec3_normalize(&normcent, &cent);
				vec3_sub(&ab, b, a);
				vec3_sub(&ac, c, a);
				vec3_cross(&norm, &ab, &ac);
				vec3_normalize(&norm, &norm);
				vec3_sub(&cent, &cent, &light);
				if (vec3_dot(&normcent, &norm) >= 0)
					continue;
				float lum = -vec3_dot(&normcent, &norm) * 255;
				// float lum = 255;
				fillTriangle(lum, &screenVerts[i], &screenVerts[i + 1], &screenVerts[i + 2]);
			}
			memcpy(middle_buffer, frame_buffer, WIDTH * HEIGHT * sizeof(uint32_t));
			free(screenVerts);
			free(triVerticies);
			delete clipped;
		}
	}
}

int main() {
	top = (plane){0, -tanf(FOV / 2 * M_PI / 180), FAR / (FAR - NEAR), -(FAR * NEAR) / (FAR - NEAR)};
	bottom = (plane){0, tanf(FOV / 2 * M_PI / 180), FAR / (FAR - NEAR), -(FAR * NEAR) / (FAR - NEAR)};
	left = (plane){ASPECT * tanf(FOV / 2 * M_PI / 180), 0, FAR / (FAR - NEAR), -(FAR * NEAR) / (FAR - NEAR)};
	right = (plane){-ASPECT * tanf(FOV / 2 * M_PI / 180), 0, FAR / (FAR - NEAR), -(FAR * NEAR) / (FAR - NEAR)};
	near = (plane){0, 0, 1, -NEAR};
	far = (plane){0, 0, 1, -FAR};
	project(proj, FOV, ASPECT, NEAR, FAR);
	if (parseFile())
		return 1;
	w = SDL_CreateWindow("Thing", SDL_WINDOWPOS_CENTERED, SDL_WINDOWPOS_CENTERED, WIDTH, HEIGHT, SDL_WINDOW_BORDERLESS | SDL_WINDOW_FULLSCREEN);
	s = SDL_GetWindowSurface(w);
	frame_buffer = (uint32_t *)calloc(WIDTH * HEIGHT, sizeof(uint32_t));
	z_buffer = (float *)calloc(WIDTH * HEIGHT, sizeof(float));
	middle_buffer = (uint32_t *)calloc(WIDTH * HEIGHT, sizeof(uint32_t));
	pthread_mutex_unlock(&paint_mutex);
	pthread_create(&t, NULL, renderLoop, NULL);
	while (true) {
		// Listen for events
		SDL_Event e;
		while (SDL_PollEvent(&e)) {
			if (e.type == SDL_QUIT)
				exit(0);
			if (e.type == SDL_KEYDOWN)
				if (e.key.keysym.sym == SDLK_ESCAPE)
					exit(0);
				else if (e.key.keysym.sym == SDLK_w)
					camera.z += 0.1;
				else if (e.key.keysym.sym == SDLK_s)
					camera.z -= 0.1;
				else if (e.key.keysym.sym == SDLK_a)
					camera.x -= 0.1;
				else if (e.key.keysym.sym == SDLK_d)
					camera.x += 0.1;
				else if (e.key.keysym.sym == SDLK_q)
					camera.y -= 0.1;
				else if (e.key.keysym.sym == SDLK_e)
					camera.y += 0.1;
				else if (e.key.keysym.sym == SDLK_r)
					raytrace = !raytrace;
		}
		printf("%f %f %f\n", camera.x, camera.y, camera.z);
		// Update screen
		long long time = SDL_GetTicks();
		SDL_LockSurface(s);
		memcpy(s->pixels, middle_buffer, WIDTH * HEIGHT * sizeof(uint32_t));
		SDL_UnlockSurface(s);
		SDL_UpdateWindowSurface(w);
		pthread_mutex_unlock(&paint_mutex);
		frame++;
		long long diff = SDL_GetTicks() - time;
		if (diff < 1000 / FPS)
			SDL_Delay(1000 / FPS - diff);
	}
	return 0;
}
