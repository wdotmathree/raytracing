#include "hip/hip_runtime.h"
#include "transform.hpp"

special void translate(float out[4][4], float x, float y, float z) {
	memset(out, 0, sizeof(float) * 16);
	out[0][0] = 1;
	out[1][1] = 1;
	out[2][2] = 1;
	out[3][3] = 1;
	out[0][3] = x;
	out[1][3] = y;
	out[2][3] = z;
}

special void rotateX(float out[4][4], float x) {
	x = x * M_PI / 180;
	memset(out, 0, sizeof(float) * 16);
	out[0][0] = 1;
	out[1][1] = cos(x);
	out[1][2] = -sin(x);
	out[2][1] = sin(x);
	out[2][2] = cos(x);
	out[3][3] = 1;
}

special void rotateY(float out[4][4], float y) {
	y = y * M_PI / 180;
	memset(out, 0, sizeof(float) * 16);
	out[0][0] = cos(y);
	out[0][2] = sin(y);
	out[1][1] = 1;
	out[2][0] = -sin(y);
	out[2][2] = cos(y);
	out[3][3] = 1;
}

special void rotateZ(float out[4][4], float z) {
	z = z * M_PI / 180;
	memset(out, 0, sizeof(float) * 16);
	out[0][0] = cos(z);
	out[0][1] = -sin(z);
	out[1][0] = sin(z);
	out[1][1] = cos(z);
	out[2][2] = 1;
	out[3][3] = 1;
}

special void scale(float out[4][4], float x, float y, float z) {
	memset(out, 0, sizeof(float) * 16);
	out[0][0] = x;
	out[1][1] = y;
	out[2][2] = z;
	out[3][3] = 1;
}

special void project(float out[4][4], float fov, float aspect, float near, float far) {
	float f = 1 / tan(fov / 2 * M_PI / 180);
	float q = far / (far - near);
	memset(out, 0, sizeof(float) * 16);
	out[0][0] = aspect * f;
	out[1][1] = f;
	out[2][2] = q;
	out[2][3] = -near * q;
	out[3][3] = 1;
}

special void to_screen(float3 *out, float3 *in) {
	out->x = (in->x / in->z + 1) * WIDTH / 2;
	out->y = (in->y / in->z + 1) * HEIGHT / 2;
	out->z = in->z;
	out->x = fminf(fmaxf(out->x, 0), WIDTH - 1);
	out->y = HEIGHT - 1 - fminf(fmaxf(out->y, 0), HEIGHT - 1);
}
